#include "hip/hip_runtime.h"
// aes_cuda.cu
// CUDA实现的AES-128 CBC加密核心

#include "aes_cuda.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

// AES S-box
__device__ __constant__ uint8_t sbox[256] = {
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76,
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0,
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15,
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75,
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84,
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf,
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8,
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2,
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73,
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb,
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79,
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08,
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a,
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e,
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf,
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16
};

__device__ uint8_t gmul(uint8_t a, uint8_t b) {
    uint8_t p = 0;
    for (int i = 0; i < 8; i++) {
        if (b & 1) p ^= a;
        uint8_t hi_bit = a & 0x80;
        a <<= 1;
        if (hi_bit) a ^= 0x1b;
        b >>= 1;
    }
    return p;
}

__device__ void sub_bytes(uint8_t* state) {
    for (int i = 0; i < 16; i++) state[i] = sbox[state[i]];
}

__device__ void shift_rows(uint8_t* state) {
    uint8_t tmp;
    // row 1
    tmp = state[1];
    state[1] = state[5]; state[5] = state[9]; state[9] = state[13]; state[13] = tmp;
    // row 2
    tmp = state[2]; state[2] = state[10]; state[10] = tmp;
    tmp = state[6]; state[6] = state[14]; state[14] = tmp;
    // row 3
    tmp = state[3]; state[3] = state[15]; state[15] = state[11]; state[11] = state[7]; state[7] = tmp;
}

__device__ void mix_columns(uint8_t* state) {
    uint8_t tmp[4];
    for (int i = 0; i < 4; i++) {
        tmp[0] = state[i*4]; tmp[1] = state[i*4+1]; tmp[2] = state[i*4+2]; tmp[3] = state[i*4+3];
        state[i*4]   = gmul(0x02, tmp[0]) ^ gmul(0x03, tmp[1]) ^ tmp[2] ^ tmp[3];
        state[i*4+1] = tmp[0] ^ gmul(0x02, tmp[1]) ^ gmul(0x03, tmp[2]) ^ tmp[3];
        state[i*4+2] = tmp[0] ^ tmp[1] ^ gmul(0x02, tmp[2]) ^ gmul(0x03, tmp[3]);
        state[i*4+3] = gmul(0x03, tmp[0]) ^ tmp[1] ^ tmp[2] ^ gmul(0x02, tmp[3]);
    }
}

__device__ void add_round_key(uint8_t* state, const uint8_t* roundKey) {
    for (int i = 0; i < 16; i++) state[i] ^= roundKey[i];
}

__device__ void key_expansion(const uint8_t* key, uint8_t* roundKeys) {
    // 只支持AES-128
    memcpy(roundKeys, key, 16);
    uint8_t temp[4];
    int i = 16, rcon = 1;
    while (i < 176) {
        for (int j = 0; j < 4; j++) temp[j] = roundKeys[i-4+j];
        if (i % 16 == 0) {
            uint8_t t = temp[0];
            temp[0] = sbox[temp[1]] ^ rcon; rcon = gmul(rcon, 2);
            temp[1] = sbox[temp[2]];
            temp[2] = sbox[temp[3]];
            temp[3] = sbox[t];
        }
        for (int j = 0; j < 4; j++) roundKeys[i] = roundKeys[i-16] ^ temp[j], i++;
    }
}

__device__ void aes128_encrypt_block(uint8_t* state, const uint8_t* roundKeys) {
    add_round_key(state, roundKeys);
    for (int round = 1; round < 10; round++) {
        sub_bytes(state);
        shift_rows(state);
        mix_columns(state);
        add_round_key(state, roundKeys + round*16);
    }
    sub_bytes(state);
    shift_rows(state);
    add_round_key(state, roundKeys + 160);
}

__global__ void aes_cbc_encrypt_kernel(const uint8_t* in, uint8_t* out, int num_blocks, const uint8_t* key, const uint8_t* iv) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_blocks) {
        uint8_t roundKeys[176];
        key_expansion(key, roundKeys);
        uint8_t block[16];
        for (int i = 0; i < 16; i++) block[i] = in[idx*16+i];
        if (idx == 0) for (int i = 0; i < 16; i++) block[i] ^= iv[i];
        else for (int i = 0; i < 16; i++) block[i] ^= out[(idx-1)*16+i];
        aes128_encrypt_block(block, roundKeys);
        for (int i = 0; i < 16; i++) out[idx*16+i] = block[i];
    }
}

extern "C" __declspec(dllexport)
void aes_encrypt_cbc(const uint8_t* in, uint8_t* out, int length, const uint8_t* key, const uint8_t* iv) {
    int num_blocks = length / 16;
    uint8_t *d_in, *d_out, *d_key, *d_iv;
    hipMalloc(&d_in, length);
    hipMalloc(&d_out, length);
    hipMalloc(&d_key, 16);
    hipMalloc(&d_iv, 16);
    hipMemcpy(d_in, in, length, hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, 16, hipMemcpyHostToDevice);
    hipMemcpy(d_iv, iv, 16, hipMemcpyHostToDevice);
    int threads = 256;
    int blocks = (num_blocks + threads - 1) / threads;
    aes_cbc_encrypt_kernel<<<blocks, threads>>>(d_in, d_out, num_blocks, d_key, d_iv);
    hipMemcpy(out, d_out, length, hipMemcpyDeviceToHost);
    hipFree(d_in); hipFree(d_out); hipFree(d_key); hipFree(d_iv);
}

// TODO: 实现完整AES-128 CBC加密 